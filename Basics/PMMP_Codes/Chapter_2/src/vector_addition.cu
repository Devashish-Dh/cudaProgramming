#include "hip/hip_runtime.h"
#include <iostream>

__global__ // device code i.e. the "kernel" that will run on the GPU
void vecAddKernel ( float* A, float* B, float* C, int n )
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

// the func that adds the vectors traditionally with a loop on the CPU :
void vecAdd ( float* A_h, float* B_h, float* C_h, int n )
{
    for ( int i = 0; i < n; i++ ){
        C_h[i] = A_h[i] + B_h[i];
    }
}

// host code
void deviceVecAdd ( float* A_h, float* B_h, float* C_h, int n )
{
    int size = n * sizeof( float ); // calculating the memory space needed on the device for our vectors

    float *A_d, *B_d, *C_d; //memory addresses of device (CPU cannot de-reference them)

    hipMalloc( (void**) &A_d, size); // tells device to allocate "size" amount of space and address it using the "A_d"
    hipMalloc( (void**) &B_d, size); //vector B

    hipMalloc( (void**) &C_d, size); // output will be stored here on the device

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice); // to send the vectors FROM cpu TO device
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // call to the kernel on the device BY THE cpu
    vecAddKernel <<< ceil(n/256.0), 256 >>>(A_d, B_d, C_d, n); // this has the <<< >>> that tells the device how many threads per block and how many blocks to use 

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost); // getting the calculated results FROM device TO host

    // very important to free device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}



int main()
{
int n = 5; 
// input vectors
float a[n] = {1,2,3,4,5};
float b[n] = {6,7,8,9,10};

// vector to hold results of the sum
float c[n];
float c_from_d[n];

vecAdd(a, b, c, n); // cpu does this

std::cout<<" CPU summed C["<<n<<"] : ";
for (int i=0; i<n; i++){
    std::cout<<" "<<c[i]<<" ";
}
std::cout<<"\n";



deviceVecAdd(a, b, c_from_d, n); //gpu does this 

std::cout<<" GPU summed C_from_d["<<n<<"] : ";
for (int i=0; i<n; i++){
    std::cout<<" "<<c_from_d[i]<<" ";
}
std::cout<<"\n";



return 0;
}
